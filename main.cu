#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576
#define NSTEP 1000
#define NKERNEL 20

int data[DATA_SIZE];

int main(){
    if(!InitCUDA()) {
        return  0;
    }
    printf( " CUDA initialized.\n ");

        //Creation of CUDA Graph
    bool graphCreated=false;
    hipGraph_t graph;
    hipGraphExec_t instance;
    
    for(int istep=0; istep<NSTEP; istep++){
        if(!graphCreated){
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
                shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated=true;
        }
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
    }

    return  0;
}

// function tests if the machine has a CUDA support
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if(count ==  0) {
        fprintf(stderr,  " There is no device.\n ");
        return  false;
    }
    int i;
    for(i =  0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >=  1) {
                break;
            }
        }
    }
    if(i == count) {
        fprintf(stderr,  " There is no device supporting CUDA 1.x.\n ");
        return  false;
    }
    hipSetDevice(i);
    return  true;
}

void GenerateNumbers( int *number,  int size)
{
     for( int i =  0; i < size; i++) {
        number[i] = rand() %  10;
    }
}
